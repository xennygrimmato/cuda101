#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/*
 * Function to perform error checking in CUDA.
 * CUDA function calls must be passed as a parameter to gpuErrchk to print Error strings, if any.
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

const int N = 128;

/*
 * CUDA Kernel
 * __global__ allows function to be called from Host (CPU), and executed in the Device (GPU).
 */
__global__ void f(int *dev_a) {
    unsigned int tid = threadIdx.x; //
    
    if(tid < N) {
        dev_a[tid] = tid * tid;
    }
}

int main(int argc, char **argv) {
    int host_a[N];
    int *dev_a;
    gpuErrchk( hipMalloc((void**)&dev_a, N * sizeof(int)));
    for(int i = 0 ; i < N ; i++) {
        host_a[i] = i;
    }
    gpuErrchk(hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice));

    /*
     * Kernel function call
     * <<< X, Y >>> : X = Number of blocks per Grid,
     *                Y = Number of threads per block
     */
    f<<<1, N>>>(dev_a);
    
    gpuErrchk(hipMemcpy(host_a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost));
    
    for(int i = 0 ; i < N ; i++) {
        printf("%d ", host_a[i]);
    }
}
