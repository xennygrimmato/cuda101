#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

const int N = 128;

__global__ void f(int *dev_a) {
    unsigned int tid = threadIdx.x;
    
    if(tid < N) {
        dev_a[tid] = tid * tid;
    }
}

int main(void) {
    
    int host_a[N];
    int *dev_a;
    gpuErrchk( hipMalloc((void**)&dev_a, N * sizeof(int)));
    for(int i = 0 ; i < N ; i++) {
        host_a[i] = i;
    }
    gpuErrchk(hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice));
    f<<<1, N>>>(dev_a);
    
    gpuErrchk(hipMemcpy(host_a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost));
    
    for(int i = 0 ; i < N ; i++) {
        printf("%d ", host_a[i]);
    }
}
